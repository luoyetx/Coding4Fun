#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char *argv[]) {
    int N = 1<<20;
    float *h_x, *h_y, *d_x, *d_y;
    h_x = (float*)malloc(N*sizeof(float));
    h_y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    
    for (int i=0; i<N; i++) {
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }

    hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<(N+256)/256, 256>>>(N, 2.0, d_x, d_y);

    hipMemcpy(h_y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i=0; i<N; i++) {
        maxError = max(maxError, abs(h_y[i]-4.0f));
    }
    printf("Max Error: %f\n", maxError);
    return 0;
}

